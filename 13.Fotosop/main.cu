#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <string>
#include <hip/hip_runtime_api.h>
#include <math.h>

// Greyscale Kernel
__global__ void greyscaleKernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        unsigned char r = input[3 * idx];
        unsigned char g = input[3 * idx + 1];
        unsigned char b = input[3 * idx + 2];
        output[idx] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

// Contrast Kernel
__global__ void contrastKernel(unsigned char* input, unsigned char* output, int width, int height, float alpha) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        for (int c = 0; c < 3; ++c) {
            int pixel = input[3 * idx + c];
            pixel = alpha * (pixel - 128) + 128;
            output[3 * idx + c] = min(max(pixel, 0), 255);
        }
    }
}

// Saturation Kernel
__global__ void saturationKernel(unsigned char* input, unsigned char* output, int width, int height, float alpha) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        float r = input[3 * idx] / 255.0f;
        float g = input[3 * idx + 1] / 255.0f;
        float b = input[3 * idx + 2] / 255.0f;
        float grey = 0.299f * r + 0.587f * g + 0.114f * b;
        r = grey + alpha * (r - grey);
        g = grey + alpha * (g - grey);
        b = grey + alpha * (b - grey);
        output[3 * idx] = min(max(r * 255.0f, 0.0f), 255.0f);
        output[3 * idx + 1] = min(max(g * 255.0f, 0.0f), 255.0f);
        output[3 * idx + 2] = min(max(b * 255.0f, 0.0f), 255.0f);
    }
}

// Greyscale Function
void greyscale(const cv::Mat& input, cv::Mat& output) {
    int width = input.cols;
    int height = input.rows;
    size_t size = width * height * sizeof(unsigned char);
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, input.total() * input.elemSize());
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input.data, input.total() * input.elemSize(), hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    greyscaleKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    hipMemcpy(output.data, d_output, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}

// Contrast Function
void contrast(const cv::Mat& input, cv::Mat& output, float alpha) {
    int width = input.cols;
    int height = input.rows;
    size_t size = input.total() * input.elemSize();
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input.data, size, hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    contrastKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, alpha);
    hipMemcpy(output.data, d_output, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}

// Saturation Function
void saturation(const cv::Mat& input, cv::Mat& output, float alpha) {
    int width = input.cols;
    int height = input.rows;
    size_t size = input.total() * input.elemSize();
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, input.data, size, hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    saturationKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, alpha);
    hipMemcpy(output.data, d_output, size, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " /path/to/file type (1: saturation, 2: contrast, 3: greyscale)" << std::endl;
        return -1;
    }
    std::string filePath = argv[1];
    int type = std::stoi(argv[2]);
    cv::Mat input = cv::imread(filePath);
    if (input.empty()) {
        std::cerr << "Error loading image" << std::endl;
        return -1;
    }
    switch (type) {
        case 3: {
            cv::Mat greyOutput(input.rows, input.cols, CV_8UC1);
            greyscale(input, greyOutput);
            cv::imwrite("./output_greyscale.jpg", greyOutput);
            break;
        }
        case 2: {
            cv::Mat contrastOutput(input.rows, input.cols, CV_8UC3);
            float alpha = 1.5f; //value
            if(argv[3] != NULL){
                alpha = std::stof(argv[3]);
            }
            contrast(input, contrastOutput, alpha);
            cv::imwrite("./output_contrast.jpg", contrastOutput);
            break;
        }
        case 1: {
            cv::Mat saturationOutput(input.rows, input.cols, CV_8UC3);
            float alpha = 1.5f; // value
            if(argv[3] != NULL){
                alpha = std::stof(argv[3]);
            }
            saturation(input, saturationOutput, alpha);
            cv::imwrite("./output_saturation.jpg", saturationOutput);
            break;
        }
        default: {
            std::cerr << "Unknown type: " << type << std::endl;
            return -1;
        }
    }
    return 0;
}